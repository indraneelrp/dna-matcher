#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"

struct MatchResultDevice {
	char sample_name[64];
	char signature_name[64];
	double match_score;
	int integrity_hash;
};

__device__ void deviceStrncpy(char* dest, const char* src, size_t n) {
	for (size_t i = 0; i < n; i++) {
		dest[i] = src[i];
		if (src[i] == '\0') break;
	}
	if (n > 0) dest[n-1] = '\0';
}

__global__ void matcherKernel(char **sample_names, int *sample_name_lens, char **sample_seqs, int *sample_seq_lens, char **sample_quals, char **sig_names, int *sig_name_lens, char **sig_seqs, int *sig_seq_lens, MatchResultDevice *results, int *match_count, int numSamples, int numSigs) {
	int sampleIdx = blockIdx.x;
	int sigIdx = threadIdx.x;

	if (sampleIdx >= numSamples || sigIdx >= numSigs) return;

	char *sample_name = sample_names[sampleIdx];
	// int sample_name_len = sample_name_lens[sampleIdx];
	char *sample_seq = sample_seqs[sampleIdx];
	int sample_seq_len = sample_seq_lens[sampleIdx];
	char *sample_qual = sample_quals[sampleIdx];

	char *sig_name = sig_names[sigIdx];
	// int sig_name_len = sig_name_lens[sigIdx];
	char *sig_seq = sig_seqs[sigIdx];
	int sig_seq_len = sig_seq_lens[sigIdx];

	bool match = false;
	double best_match_score = 0;
	for (int i = 0; i < sample_seq_len - sig_seq_len; i++) {
		int j;
		for (j = 0; j < sig_seq_len; j++) {
			if (sample_seq[i+j] != 'N' && sig_seq[j] != 'N' && sample_seq[i+j] != sig_seq[j]) break;
		}
		if (j == sig_seq_len) {
			match = true;
			double curr_match_score = 0;
			for (int k = i; k < i+j; k++) {
				double k_phred_score = 0;
				if (sample_seq[k] != 'N') {
					k_phred_score = double(sample_qual[k]) - 33;
				}
				curr_match_score += k_phred_score;
			}
			curr_match_score /= sig_seq_len;
			if (curr_match_score >= best_match_score) {
				best_match_score = curr_match_score;
			}
		}
	}
	
	if (match == true) {
		int sample_integrity_hash = 0;
		for (int i = 0; i < sample_seq_len; i++) {		
			double i_phred_score = 0;
			if (sample_seq[i] != 'N') {
				i_phred_score = double(sample_qual[i]) - 33;
			}
			sample_integrity_hash += i_phred_score;
		}
		sample_integrity_hash %= 97;

		int result_idx = atomicAdd(match_count, 1);
		deviceStrncpy(results[result_idx].sample_name, sample_name, 64);
		deviceStrncpy(results[result_idx].signature_name, sig_name, 64);
		results[result_idx].match_score = best_match_score;
		results[result_idx].integrity_hash = sample_integrity_hash;
	}
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
	// this is the "main" fn that we run the device fn from
	// set number of blocks per grid and number of threads per block
	int numSamples = samples.size();	// blocks per grid
	int numSigs = signatures.size();
	int blocksPerGrid = 2200;
	//int threadsPerBlock = 1024;
	int threadsPerBlock = ((numSigs+31)/32) * 32;

	// allocate arrays for pointers
	char **h_sample_names = new char*[numSamples];
	int *h_sample_name_lens = new int[numSamples];
	char **h_sample_seqs = new char*[numSamples];
	int *h_sample_seq_lens = new int[numSamples];
	char **h_sample_quals = new char*[numSamples];

	char **h_sig_names = new char*[numSigs];
	int *h_sig_name_lens = new int[numSigs];
	char **h_sig_seqs = new char*[numSigs];
	int *h_sig_seq_lens = new int[numSigs];

	// allocate memory for GPU (samples, sigs: name, name_len, seq, seq_len, qual. flattened result array). 
	char **d_sample_names, **d_sample_seqs, **d_sample_quals, **d_sig_names, **d_sig_seqs;
	int *d_sample_name_lens, *d_sample_seq_lens, *d_sig_name_lens, *d_sig_seq_lens, *d_match_count;
	MatchResultDevice *d_results;

	hipMalloc(&d_sample_names, numSamples * sizeof(char *));
	hipMalloc(&d_sample_seqs, numSamples * sizeof(char *));
	hipMalloc(&d_sample_quals, numSamples * sizeof(char *));
	hipMalloc(&d_sample_name_lens, numSamples * sizeof(int));
	hipMalloc(&d_sample_seq_lens, numSamples * sizeof(int));

	hipMalloc(&d_sig_names, numSigs * sizeof(char *));
	hipMalloc(&d_sig_seqs, numSigs * sizeof(char *));
	hipMalloc(&d_sig_name_lens, numSigs * sizeof(int));
	hipMalloc(&d_sig_seq_lens, numSigs * sizeof(int));

	hipMalloc(&d_results, numSamples * numSigs * sizeof(MatchResultDevice));
	hipMalloc(&d_match_count, sizeof(int));


	// copy samples and signatures to GPU
	for (int i = 0; i < numSamples; i++) {
		h_sample_name_lens[i] = samples[i].name.size();
		h_sample_seq_lens[i] = samples[i].seq.size();

		hipMalloc(&h_sample_names[i], h_sample_name_lens[i] * sizeof(char));
		hipMemcpy(h_sample_names[i], samples[i].name.c_str(), h_sample_name_lens[i] * sizeof(char), hipMemcpyHostToDevice);

		hipMalloc(&h_sample_seqs[i], h_sample_seq_lens[i] * sizeof(char));
		hipMemcpy(h_sample_seqs[i], samples[i].seq.c_str(), h_sample_seq_lens[i] * sizeof(char), hipMemcpyHostToDevice);

		hipMalloc(&h_sample_quals[i], h_sample_seq_lens[i] * sizeof(char));
		hipMemcpy(h_sample_quals[i], samples[i].qual.c_str(), h_sample_seq_lens[i] * sizeof(char), hipMemcpyHostToDevice);
	}

	for (int i = 0; i < numSigs; i++) {
		h_sig_name_lens[i] = signatures[i].name.size();
		h_sig_seq_lens[i] = signatures[i].seq.size();

		hipMalloc(&h_sig_names[i], h_sig_name_lens[i] * sizeof(char));
		hipMemcpy(h_sig_names[i], signatures[i].name.c_str(), h_sig_name_lens[i] * sizeof(char), hipMemcpyHostToDevice);

		hipMalloc(&h_sig_seqs[i], h_sig_seq_lens[i] * sizeof(char));
		hipMemcpy(h_sig_seqs[i], signatures[i].seq.c_str(), h_sig_seq_lens[i] * sizeof(char), hipMemcpyHostToDevice);
	}


	// copy pointers to GPU
	hipMemcpy(d_sample_names, h_sample_names, numSamples * sizeof(char *), hipMemcpyHostToDevice);
	hipMemcpy(d_sample_name_lens, h_sample_name_lens, numSamples * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sample_seqs, h_sample_seqs, numSamples * sizeof(char *), hipMemcpyHostToDevice);
	hipMemcpy(d_sample_seq_lens, h_sample_seq_lens, numSamples * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sample_quals, h_sample_quals, numSamples * sizeof(char *), hipMemcpyHostToDevice);


	hipMemcpy(d_sig_names, h_sig_names, numSigs * sizeof(char *), hipMemcpyHostToDevice);
	hipMemcpy(d_sig_name_lens, h_sig_name_lens, numSigs * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sig_seqs, h_sig_seqs, numSigs * sizeof(char *), hipMemcpyHostToDevice);
	hipMemcpy(d_sig_seq_lens, h_sig_seq_lens, numSigs * sizeof(int), hipMemcpyHostToDevice);
	
	int zero = 0;
	hipMemcpy(d_match_count, &zero, sizeof(int), hipMemcpyHostToDevice);


	// run kernel function then cudeDeviceSynchronize()
	matcherKernel<<<blocksPerGrid, threadsPerBlock>>>(d_sample_names, d_sample_name_lens, d_sample_seqs, d_sample_seq_lens, d_sample_quals, d_sig_names, d_sig_name_lens, d_sig_seqs, d_sig_seq_lens, d_results, d_match_count, numSamples, numSigs);
	hipDeviceSynchronize();

	int h_match_count;
	hipMemcpy(&h_match_count, d_match_count, sizeof(int), hipMemcpyDeviceToHost);


	// copy results from kernel function into matches
	MatchResultDevice *h_results = new MatchResultDevice[h_match_count];
	hipMemcpy(h_results, d_results, h_match_count * sizeof(MatchResultDevice), hipMemcpyDeviceToHost);


	// process results
	matches.clear();
	for (int i = 0; i < h_match_count; i++) {
		matches.push_back({
				std::string(h_results[i].sample_name),
				std::string(h_results[i].signature_name),
				h_results[i].match_score,
				h_results[i].integrity_hash });
	}


	// free memory
	for (int i = 0; i < numSamples; i++) {
		hipFree(h_sample_names[i]);
		hipFree(h_sample_seqs[i]);
		hipFree(h_sample_quals[i]);
	}
	for (int j = 0; j < numSigs; j++) {
		hipFree(h_sig_names[j]);
		hipFree(h_sig_seqs[j]);
	}
	delete[] h_sample_names;
	delete[] h_sample_name_lens;
	delete[] h_sample_seqs;
	delete[] h_sample_seq_lens;
	delete[] h_sample_quals;
	delete[] h_sig_names;
	delete[] h_sig_name_lens;
	delete[] h_sig_seqs;
	delete[] h_sig_seq_lens;;
	delete[] h_results;

	hipFree(d_sample_names);
	hipFree(d_sample_name_lens);
	hipFree(d_sample_seqs);
	hipFree(d_sample_seq_lens);
	hipFree(d_sample_quals);
	hipFree(d_sig_names);
	hipFree(d_sig_name_lens);
	hipFree(d_sig_seqs);
	hipFree(d_sig_seq_lens);
	hipFree(d_results);
	hipFree(d_match_count);
}
